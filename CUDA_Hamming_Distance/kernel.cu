#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <ctime> 
#include <fstream>
#include <string>
#include<iostream>

#define SIZE 1000
#define SEQUENCES 100
#define BLOCK 32
#define THREATS 1024

//__global__ void sum(int* d_sum, int* d_data)
//{
//	extern __shared__ float temp[];
//	int tid = threadIdx.x;
//	temp[tid] = d_data[tid + blockIdx.x * blockDim.x];
//	for (int d = blockDim.x >> 1; d >= 1; d >>= 1) {
//		__syncthreads();
//		if (tid < d) temp[tid] += temp[tid + d];
//	}
//	if (tid == 0) d_sum[blockIdx.x] = temp[0];
//}

__global__ void HammingDistance(int *c, const int* a, const int* b ,long const int* size)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (; i < *size; i += stride) {
		//atomicAdd(c, a[i] != b[i]);
		//atomicAdd(c, a[i] ^ b[i]);
		a[i] != b[i] ? atomicAdd(c, 1) : 1;
	}

}

int* RandBinSeq(int n) {
	int* result = new int[n];

	for (int i = 0; i < n; i++) {
		result[i] = rand() % 2;
	}
	return result;
}

int** Many(int size, int num) {
	int** result = new int*[num];
	for (int i = 0; i < num; i++) {
		result[i] = RandBinSeq(size);
	}
	return result;
}

void writeToFile() {
	std::ofstream file;
	file.open("sequences.txt", std::ios::out);
	if (file.good() == true)
	{
		for (int i = 0; i < SEQUENCES; i++) {
			file << "[";
			for (int j = 0; j < SIZE; j++) {
				file << rand() % 2;
			}
			file << "]\n";
			file.flush();
		}
		file.close();
	}
	printf("Save in file\n");
}

int** readFromFile() {
	std::ifstream file;
	file.open("sequences.txt", std::ios::in);
	//char line [SIZE + 3] ;
	std::string line;
	int** result = new int* [SEQUENCES];
	int i = 0, l = 0;
	if (file.good() == true)
	{
		while (l<SEQUENCES) {
			int *r = new int[SIZE];
			getline(file, line);
			for (i = 1; line[i] != ']'; i++) {
				r[i-1] = (line[i] - '0');
			}
			result[l++] = r;
		}
		file.close();
	}
	printf("Load from file\n");
	return result;
}


int main()
{
	srand(time(NULL));
	//srand(1);
	//writeToFile();
	int** seq = readFromFile();
	long  int* size = (long int*)malloc(sizeof(long int));
	*size = SIZE;
	int* c = new int[SEQUENCES * (SEQUENCES - 1)];


	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	long int* sizeC = 0;
	hipError_t cudaStatus;

	//cudaStatus = hipMalloc((void**)&dev_c, sizeof(int));
	cudaStatus = hipMalloc((void**)&dev_c, (SEQUENCES * (SEQUENCES - 1)) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, *size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, *size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&sizeC, sizeof(long int) * SEQUENCES);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(sizeC, size, sizeof(long int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy b failed!");
		goto Error;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Launch a kernel on the GPU with one thread for each element.
	hipEventRecord(start);
	int k = 0;
	for (int i = 0; i < SEQUENCES-1; i++) {
		for (int j = i+1; j < SEQUENCES; j++,k++) {
			cudaStatus = hipMemcpy(dev_a, seq[i], *size * sizeof(int), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy a failed!");
				goto Error;
			}

			cudaStatus = hipMemcpy(dev_b, seq[j], *size * sizeof(int), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy b failed!");
				goto Error;
			}

			//*c = 0;
			//cudaStatus = hipMemcpy(dev_c, c, sizeof(int), hipMemcpyHostToDevice);
			//if (cudaStatus != hipSuccess) {
			//	fprintf(stderr, "hipMemcpy b failed!");
			//	goto Error;
			//}
			 

			HammingDistance <<< BLOCK, THREATS >>> (dev_c+k, dev_a, dev_b, sizeC);
			//HammingDistance <<< BLOCK, THREATS >>> (dev_c, dev_a, dev_b, sizeC);

			//cudaStatus = hipMemcpy(c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
			//if (cudaStatus != hipSuccess) {
			//	fprintf(stderr, "hipMemcpy c failed!");
			//	goto Error;
			//}
			//printf("The Hamming distance between %d and %d seqence is %d. ", i, j, *c);
			//if (*c == 1)printf("Pair with distance equal 1");
			//printf("\n");
		}
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching HammingDistance!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, sizeof(int) * SEQUENCES * (SEQUENCES - 1), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy c failed!");
		goto Error;
	}
	k = 0;
	int s = 0;
	for (int i = 0; i < SEQUENCES - 1; i++) {
		for (int j = i+1; j < SEQUENCES; j++,k++) {
			if (c[k] == 1) {
				s++;
				printf("Pair with distance equal 1. ");
				printf("The Hamming distance between %d and %d seqence is %d.\n", i, j, *(c + k));
			}
		}
	}
	printf("There is %d pairs with the Hamming distance equal 1\n", s);


	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Time ms %f \n\n\n",milliseconds);

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	delete(c);
	delete(size);
	for (int i = 0; i < SEQUENCES; i++) {
		delete(seq[i]);
	}
	delete(seq);

    return 0;
}